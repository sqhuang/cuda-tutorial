#include "hip/hip_runtime.h"
#include<iostream>
#include "utils.h"

template <class T>
static void __global__ _cuda_relu(const T* a, unsigned total, T* b)
{
    const unsigned tid = threadIdx.x + blockDim.x * blockIdx.x;
    const unsigned skip = blockDim.x * gridDim.x;
    for (unsigned i = tid; i < total; i += skip)
    {
        T v = a[i];
        b[i] = max(v, T(0));
    }
}


int main() {
    unsigned int size = 100;
    float* input  = (float*)malloc(size * sizeof(float));
    float* output = (float*)malloc(size * sizeof(float));
    for(int i = 0; i < size; i++){
        input[i] = rand()/double(RAND_MAX) - 0.5;
    }

    // malloc
    float* pinput;
    CUDA_CHECK(hipMalloc(&pinput,  size * sizeof(float)));
    float* poutput;
    CUDA_CHECK(hipMalloc(&poutput, size * sizeof(float)));

    // copy
    CUDA_CHECK(hipMemcpy(pinput, input, size * sizeof(float), hipMemcpyHostToDevice));
    
    int blockCount  = 1;
    int threadCount = 10;

    double timestart_cpu = cpuTimerInMS();
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    CUDA_CHECK(hipEventRecord(start));

    _cuda_relu<<<blockCount, threadCount, 0, 0>>>(pinput, size, poutput);
    CUDA_CHECK(hipMemcpy(output, poutput, size*sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(start));
    CUDA_CHECK(hipEventSynchronize(stop));
    // calculate the elapsed time between two events

    float timeelaps_cuda;
    CUDA_CHECK(hipEventElapsedTime(&timeelaps_cuda, start, stop));
    double timeelaps_cpu = cpuTimerInMS() - timestart_cpu;
    std::cout<<"[CPU]   time elapsed in millisecond: "<< timeelaps_cpu <<"" << std::endl;
    std::cout<<"[CUDA]  time elapsed in millisecond: "<< timeelaps_cuda <<"" << std::endl;
//    for(int i=0; i<size; i++){
//        std::cout<<"in[" << i <<"]: "<<input[i]<<"\t out[" << i <<"]: "<<output[i]<<std::endl;
//    }
    // free
    CUDA_CHECK(hipFree(pinput));
    CUDA_CHECK(hipFree(poutput));
    free(input);
    free(output);
    return 0;
}
