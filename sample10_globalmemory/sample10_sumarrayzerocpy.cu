#include "hip/hip_runtime.h"
#include<iostream>
#include "utils.h"

void initialData(float *ip, int size)
{
    initialFloat(ip, size);
}

void sumArraysOnHost(float *A, float *B, float *C, const int N)
{
    for (int idx = 0; idx < N; idx++)
    {
        C[idx] = A[idx] + B[idx];
    }
}

__global__ void sumArrays(float *A, float *B, float *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) C[i] = A[i] + B[i];
}

__global__ void sumArraysZeroCopy(float *A, float *B, float *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) C[i] = A[i] + B[i];
}

int main(int argc, char **argv)
{
    // set up device
    int dev = 0;
    CUDA_CHECK(hipSetDevice(dev));

    // get device properties
    hipDeviceProp_t deviceProp;
    CUDA_CHECK(hipGetDeviceProperties(&deviceProp, dev));

    // check if support mapped memory
    // 测试设备是否支持固定设备映射
    if (!deviceProp.canMapHostMemory)
    {
        printf("Device %d does not support mapping CPU host memory!\n", dev);
        CUDA_CHECK(hipDeviceReset());
        exit(EXIT_SUCCESS);
    }

    printf("Using Device %d: %s ", dev, deviceProp.name);

    // set up data size of vectors
    int ipower = 10;

    if (argc > 1) ipower = atoi(argv[1]);

    int nElem = 1 << ipower;
    size_t nBytes = nElem * sizeof(float);

    if (ipower < 18)
    {
        printf("Vector size %d power %d  nbytes  %3.0f KB\n", nElem, ipower,
               (float)nBytes / (1024.0f));
    }
    else
    {
        printf("Vector size %d power %d  nbytes  %3.0f MB\n", nElem, ipower,
               (float)nBytes / (1024.0f * 1024.0f));
    }

    // part 1: using device memory
    // malloc host memory
    float *h_A, *h_B, *hostRef, *gpuRef;
    h_A     = (float *)malloc(nBytes);
    h_B     = (float *)malloc(nBytes);
    hostRef = (float *)malloc(nBytes);
    gpuRef  = (float *)malloc(nBytes);

    // initialize data at host side
    initialData(h_A, nElem);
    initialData(h_B, nElem);
    memset(hostRef, 0, nBytes);
    memset(gpuRef,  0, nBytes);

    // add vector at host side for result checks
    sumArraysOnHost(h_A, h_B, hostRef, nElem);

    // malloc device global memory
    float *d_A, *d_B, *d_C;
    CUDA_CHECK(hipMalloc((float**)&d_A, nBytes));
    CUDA_CHECK(hipMalloc((float**)&d_B, nBytes));
    CUDA_CHECK(hipMalloc((float**)&d_C, nBytes));

    // transfer data from host to device
    CUDA_CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice));

    // set up execution configuration
    int iLen = 512;
    dim3 block (iLen);
    dim3 grid  ((nElem + block.x - 1) / block.x);

    sumArrays<<<grid, block>>>(d_A, d_B, d_C, nElem);

    // copy kernel result back to host side
    CUDA_CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    // check device results
    checkResult(hostRef, gpuRef, nElem);

    // free device global memory
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));

    // free host memory
    free(h_A);
    free(h_B);

    // part 2: using zerocopy memory for array A and B
    // allocate zerocpy memory
    unsigned int flags =  hipHostMallocMapped;
    CUDA_CHECK(hipHostAlloc((void **)&h_A, nBytes, flags));
    CUDA_CHECK(hipHostAlloc((void **)&h_B, nBytes, flags));

    // initialize data at host side
    initialData(h_A, nElem);
    initialData(h_B, nElem);
    memset(hostRef, 0, nBytes);
    memset(gpuRef,  0, nBytes);

    // pass the pointer to device
    CUDA_CHECK(hipHostGetDevicePointer((void **)&d_A, (void *)h_A, 0));
    CUDA_CHECK(hipHostGetDevicePointer((void **)&d_B, (void *)h_B, 0));

    // add at host side for result checks
    sumArraysOnHost(h_A, h_B, hostRef, nElem);

    // execute kernel with zero copy memory
    sumArraysZeroCopy<<<grid, block>>>(d_A, d_B, d_C, nElem);

    // copy kernel result back to host side
    CUDA_CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    // check device results
    checkResult(hostRef, gpuRef, nElem);

    // free  memory
    CUDA_CHECK(hipFree(d_C));
    CUDA_CHECK(hipHostFree(h_A));
    CUDA_CHECK(hipHostFree(h_B));

    free(hostRef);
    free(gpuRef);

    // reset device
    CUDA_CHECK(hipDeviceReset());
    return 0;
}
