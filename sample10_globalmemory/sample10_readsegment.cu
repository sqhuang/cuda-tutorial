#include "hip/hip_runtime.h"
#include<iostream>
#include "utils.h"

void initialData(float *ip,  int size)
{
    initialFloat(ip, size);
}


void sumArraysOnHost(float *A, float *B, float *C, const int n, int offset)
{
    for (int idx = offset, k = 0; idx < n; idx++, k++)
    {
        C[k] = A[idx] + B[idx];
    }
}

__global__ void warmup(float *A, float *B, float *C, const int n, int offset)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = i + offset;

    if (k < n) C[i] = A[k] + B[k];
}

__global__ void readOffset(float *A, float *B, float *C, const int n,
                           int offset)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = i + offset;

    if (k < n) C[i] = A[k] + B[k];
}

int main(int argc, char **argv)
{
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CUDA_CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("%s starting reduction at ", argv[0]);
    printf("device %d: %s ", dev, deviceProp.name);
    CUDA_CHECK(hipSetDevice(dev));

    // set up array size
    int nElem = 1 << 20; // total number of elements to reduce
    printf(" with array size %d\n", nElem);
    size_t nBytes = nElem * sizeof(float);

    // set up offset for summary
    int blocksize = 512;
    int offset = 0;

    if (argc > 1) offset    = atoi(argv[1]);

    if (argc > 2) blocksize = atoi(argv[2]);

    // execution configuration
    dim3 block (blocksize, 1);
    dim3 grid  ((nElem + block.x - 1) / block.x, 1);

    // allocate host memory
    float *h_A = (float *)malloc(nBytes);
    float *h_B = (float *)malloc(nBytes);
    float *hostRef = (float *)malloc(nBytes);
    float *gpuRef  = (float *)malloc(nBytes);

    //  initialize host array
    initialData(h_A, nElem);
    memcpy(h_B, h_A, nBytes);

    //  summary at host side
    sumArraysOnHost(h_A, h_B, hostRef, nElem, offset);

    // allocate device memory
    float *d_A, *d_B, *d_C;
    CUDA_CHECK(hipMalloc((float**)&d_A, nBytes));
    CUDA_CHECK(hipMalloc((float**)&d_B, nBytes));
    CUDA_CHECK(hipMalloc((float**)&d_C, nBytes));

    // copy data from host to device
    CUDA_CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_A, nBytes, hipMemcpyHostToDevice));

    //  kernel 1:
    double iStart =  cpuTimerInS();
    warmup<<<grid, block>>>(d_A, d_B, d_C, nElem, offset);
    CUDA_CHECK(hipDeviceSynchronize());
    double iElaps = cpuTimerInS() - iStart;
    printf("warmup     <<< %4d, %4d >>> offset %4d elapsed %f sec\n", grid.x,
           block.x, offset, iElaps);
    CUDA_CHECK(hipGetLastError());

    iStart = cpuTimerInS() ;
    readOffset<<<grid, block>>>(d_A, d_B, d_C, nElem, offset);
    CUDA_CHECK(hipDeviceSynchronize());
    iElaps = cpuTimerInS() - iStart;
    printf("readOffset <<< %4d, %4d >>> offset %4d elapsed %f sec\n", grid.x,
           block.x, offset, iElaps);
    CUDA_CHECK(hipGetLastError());

    // copy kernel result back to host side and check device results
    CUDA_CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
    checkResult(hostRef, gpuRef, nElem - offset);

    // free host and device memory
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));
    free(h_A);
    free(h_B);

    // reset device
    CUDA_CHECK(hipDeviceReset());
    return EXIT_SUCCESS;
}