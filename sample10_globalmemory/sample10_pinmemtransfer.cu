#include<iostream>
#include "utils.h"

int main(int argc, char **argv)
{
    // set up device
    int dev = 0;
    CUDA_CHECK(hipSetDevice(dev));

    // memory size
    unsigned int isize = 1 << 22;
    unsigned int nbytes = isize * sizeof(float);

    // get device information
    hipDeviceProp_t deviceProp;
    CUDA_CHECK(hipGetDeviceProperties(&deviceProp, dev));

    if (!deviceProp.canMapHostMemory)
    {
        printf("Device %d does not support mapping CPU host memory!\n", dev);
        CUDA_CHECK(hipDeviceReset());
        exit(EXIT_SUCCESS);
    }

    printf("%s starting at ", argv[0]);
    printf("device %d: %s memory size %d nbyte %5.2fMB canMap %d\n", dev,
           deviceProp.name, isize, nbytes / (1024.0f * 1024.0f),
           deviceProp.canMapHostMemory);

    // allocate pinned host memory
    float *h_a;
    CUDA_CHECK(hipHostMalloc ((float **)&h_a, nbytes));

    // allocate device memory
    float *d_a;
    CUDA_CHECK(hipMalloc((float **)&d_a, nbytes));

    // initialize host memory
    memset(h_a, 0, nbytes);

    for (int i = 0; i < isize; i++) h_a[i] = 100.10f;

    // transfer data from the host to the device
    CUDA_CHECK(hipMemcpy(d_a, h_a, nbytes, hipMemcpyHostToDevice));

    // transfer data from the device to the host
    CUDA_CHECK(hipMemcpy(h_a, d_a, nbytes, hipMemcpyDeviceToHost));

    // free memory
    CUDA_CHECK(hipFree(d_a));
    CUDA_CHECK(hipHostFree(h_a));

    // reset device
    CUDA_CHECK(hipDeviceReset());
    return EXIT_SUCCESS;
}