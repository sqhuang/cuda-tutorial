#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include "utils.h"

/*
 * This example demonstrates submitting work to a CUDA stream in depth-first
 * order. Work submission in depth-first order may introduce false-dependencies
 * between unrelated tasks in different CUDA streams, limiting the parallelism
 * of a CUDA application. kernel_1, kernel_2, kernel_3, and kernel_4 simply
 * implement identical, dummy computation. Separate kernels are used to make the
 * scheduling of these kernels simpler to visualize in the Visual Profiler.
 */

#define N 300000
#define NSTREAM 4

__global__ void kernel_1()
{
    double sum = 0.0;

    for(int i = 0; i < N; i++)
    {
        sum = sum + tan(0.1) * tan(0.1);
    }
}

__global__ void kernel_2()
{
    double sum = 0.0;

    for(int i = 0; i < N; i++)
    {
        sum = sum + tan(0.1) * tan(0.1);
    }
}

__global__ void kernel_3()
{
    double sum = 0.0;

    for(int i = 0; i < N; i++)
    {
        sum = sum + tan(0.1) * tan(0.1);
    }
}

__global__ void kernel_4()
{
    double sum = 0.0;

    for(int i = 0; i < N; i++)
    {
        sum = sum + tan(0.1) * tan(0.1);
    }
}

int main(int argc, char **argv)
{
    int n_streams = NSTREAM;
    int isize = 1;
    int iblock = 1;
    int bigcase = 0;

    // get argument from command line
    if (argc > 1) n_streams = atoi(argv[1]);

    if (argc > 2) bigcase = atoi(argv[2]);

    float elapsed_time;

    // set up max connectioin
    // 并行硬件连接数量，每个 CUDA 流都会被映射到单一 CUDA 设备连接中。
    // 如果流的数量超过了硬件连接数量，多个流将共享一个连接。
    // 当多个流共享相同的硬件工作队列时，肯呢个会产生虚假的依赖关系。

    char* iname = "CUDA_DEVICE_MAX_CONNECTIONS";
    setenv (iname, "32", 1);
    /*
    NAME
           setenv - change or add an environmentvariable      //改变或添加一个环境变量
    SYNOPSIS
           #include <stdlib.h>
           int setenv(const char *name, const char*value, int overwrite);
               int unsetenv(const char *name);
    
    overwrite参数：
        非0表示覆盖原有环境变量，0表示不覆盖。
    */
    char *ivalue =  getenv (iname);
    printf ("%s = %s\n", iname, ivalue);

    int dev = 0;
    hipDeviceProp_t deviceProp;
    CUDA_CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("> Using Device %d: %s with num_streams=%d\n", dev, deviceProp.name,
           n_streams);
    CUDA_CHECK(hipSetDevice(dev));

    // check if device support hyper-q
    if (deviceProp.major < 3 || (deviceProp.major == 3 && deviceProp.minor < 5))
    {
        if (deviceProp.concurrentKernels == 0)
        {
            printf("> GPU does not support concurrent kernel execution (SM 3.5 "
                    "or higher required)\n");
            printf("> CUDA kernel runs will be serialized\n");
        }
        else
        {
            printf("> GPU does not support HyperQ\n");
            printf("> CUDA kernel runs will have limited concurrency\n");
        }
    }

    printf("> Compute Capability %d.%d hardware with %d multi-processors\n",
           deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);

    // Allocate and initialize an array of stream handles
    hipStream_t *streams = (hipStream_t *) malloc(n_streams * sizeof(
                                hipStream_t));

    for (int i = 0 ; i < n_streams ; i++)
    {
        CUDA_CHECK(hipStreamCreate(&(streams[i])));
    }

    // run kernel with more threads
    if (bigcase == 1)
    {
        iblock = 512;
        isize = 1 << 12;
    }

    // set up execution configuration
    dim3 block (iblock);
    dim3 grid  (isize / iblock);
    printf("> grid %d block %d\n", grid.x, block.x);

    // creat events
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    // record start event
    CUDA_CHECK(hipEventRecord(start, 0));

    // dispatch job with depth first ordering
    for (int i = 0; i < n_streams; i++)
    {
        kernel_1<<<grid, block, 0, streams[i]>>>();
        kernel_2<<<grid, block, 0, streams[i]>>>();
        kernel_3<<<grid, block, 0, streams[i]>>>();
        kernel_4<<<grid, block, 0, streams[i]>>>();
    }

    // record stop event
    CUDA_CHECK(hipEventRecord(stop, 0));
    CUDA_CHECK(hipEventSynchronize(stop));

    // calculate elapsed time
    CUDA_CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
    printf("Measured time for parallel execution = %.3fs\n",
           elapsed_time / 1000.0f);

    // release all stream
    for (int i = 0 ; i < n_streams ; i++)
    {
        CUDA_CHECK(hipStreamDestroy(streams[i]));
    }

    free(streams);

    // destroy events
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    // reset device
    CUDA_CHECK(hipDeviceReset());

    return 0;
}