#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include "utils.h"
#include <omp.h>

/*
 * An example of using OpenMP to parallelize the creation of CUDA work in
 * multiple streams. This example using n_streams OpenMP threads to launch 4
 * kernels in each stream. Note the new pragma introduced, #pragma omp parallel.
 */

#define N 300000
#define NSTREAM 4

__global__ void kernel_1()
{
    double sum = 0.0;

    for(int i = 0; i < N; i++)
    {
        sum = sum + tan(0.1) * tan(0.1);
    }
}

__global__ void kernel_2()
{
    double sum = 0.0;

    for(int i = 0; i < N; i++)
    {
        sum = sum + tan(0.1) * tan(0.1);
    }
}

__global__ void kernel_3()
{
    double sum = 0.0;

    for(int i = 0; i < N; i++)
    {
        sum = sum + tan(0.1) * tan(0.1);
    }
}

__global__ void kernel_4()
{
    double sum = 0.0;

    for(int i = 0; i < N; i++)
    {
        sum = sum + tan(0.1) * tan(0.1);
    }
}

int main(int argc, char **argv)
{
    int n_streams = NSTREAM;
    int isize = 1;
    int iblock = 1;
    int bigcase = 0;

    // get argument from command line
    if (argc > 1) n_streams = atoi(argv[1]);

    if (argc > 2) bigcase = atoi(argv[2]);

    float elapsed_time;

    // set up max connectioin
    char* iname = "CUDA_DEVICE_MAX_CONNECTIONS";
    setenv (iname, "32", 1);
    char *ivalue =  getenv (iname);
    printf ("%s = %s\n", iname, ivalue);

    int dev = 0;
    hipDeviceProp_t deviceProp;
    CUDA_CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("> Using Device %d: %s with num_streams=%d\n", dev, deviceProp.name,
           n_streams);
    CUDA_CHECK(hipSetDevice(dev));

    // check if device support hyper-q
    if (deviceProp.major < 3 || (deviceProp.major == 3 && deviceProp.minor < 5))
    {
        if (deviceProp.concurrentKernels == 0)
        {
            printf("> GPU does not support concurrent kernel execution (SM 3.5 "
                    "or higher required)\n");
            printf("> CUDA kernel runs will be serialized\n");
        }
        else
        {
            printf("> GPU does not support HyperQ\n");
            printf("> CUDA kernel runs will have limited concurrency\n");
        }
    }

    printf("> Compute Capability %d.%d hardware with %d multi-processors\n",
           deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);

    // Allocate and initialize an array of stream handles
    hipStream_t *streams = (hipStream_t *) malloc(n_streams * sizeof(
                                hipStream_t));

    for (int i = 0 ; i < n_streams ; i++)
    {
        CUDA_CHECK(hipStreamCreate(&(streams[i])));
    }

    // run kernel with more threads
    if (bigcase == 1)
    {
        iblock = 512;
        isize = 1 << 12;
    }

    // set up execution configuration
    dim3 block (iblock);
    dim3 grid  (isize / iblock);
    printf("> grid %d block %d\n", grid.x, block.x);

    // creat events
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    // record start event
    CUDA_CHECK(hipEventRecord(start, 0));

    // dispatch job with depth first ordering using OpenMP
    omp_set_num_threads(n_streams);
    // OpenMp 函数 omp_set_num_threads 用来指定在 OpenMP 并行区域里要用到的 CPU 核心的数量。
    #pragma omp parallel
    {
        int i = omp_get_thread_num();
        kernel_1<<<grid, block, 0, streams[i]>>>();
        kernel_2<<<grid, block, 0, streams[i]>>>();
        kernel_3<<<grid, block, 0, streams[i]>>>();
        kernel_4<<<grid, block, 0, streams[i]>>>();
    }
    // 编译器指令 #pragma omp parllel 将花括号之间的代码标记为并行部分。
    // omp_get_thread_num 函数为每个主机线程返回唯一一个线程 ID。
    // record stop event
    CUDA_CHECK(hipEventRecord(stop, 0));
    CUDA_CHECK(hipEventSynchronize(stop));

    // calculate elapsed time
    CUDA_CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
    printf("Measured time for parallel execution = %.3fs\n",
           elapsed_time / 1000.0f);

    // release all stream
    for (int i = 0 ; i < n_streams ; i++)
    {
        CUDA_CHECK(hipStreamDestroy(streams[i]));
    }

    free(streams);

    // destroy events
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    // reset device
    CUDA_CHECK(hipDeviceReset());

    return 0;
}
